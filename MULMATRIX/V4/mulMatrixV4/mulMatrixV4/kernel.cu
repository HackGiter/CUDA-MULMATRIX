#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include "hip/hip_texture_types.h"

#include "Header.cuh"

#include <stdio.h>

__host__ void HostToDeviceMat(const MATRIX, MATRIX*);
__host__ float MulMat(const MATRIX, const MATRIX, MATRIX*, int);
__host__ float CublasMM(const MATRIX, const MATRIX, MATRIX*);
__host__ void CompareSGEMM(float* , int);

int main()
{
   //int width = 256, height = 256;
   //float elapsetime1, elapsetime2;
   //MATRIX* m1 = (MATRIX*)malloc(sizeof(MATRIX)),
   //      * m2 = (MATRIX*)malloc(sizeof(MATRIX)),
   //      * m3 = (MATRIX*)malloc(sizeof(MATRIX)),
   //      * m4 = (MATRIX*)malloc(sizeof(MATRIX));

   //width *= 1;
   //height *= 1;
   //printf("Test Matrix : WIDTH: %d HEIGHT: %d\n", width, height);
   //InitMat(m1, width, height);
   //InitMat(m2, height, width);
   //InitMat(m3, height, height);
   //InitMat(m4, height, height);

   //SetMat(m3, 0);
   //SetMat(m4, 0);
   //HostRANDSetMat(m1, 1);
   //HostRANDSetMat(m2, 2);
   ////SetMat(m1, 1);
   ////SetMat(m2, 1.5);
   //warmup << <1, 1 >> > ();

   //hipDeviceSynchronize();
   //elapsetime1 = MulMat(*m1, *m2, m3, 2);
   //hipDeviceSynchronize();
   //elapsetime2 = CublasMM(*m1, *m2, m4);
   ////elapsetime2 = MulMat(*m1, *m2, m4, 5);
   //VerifyMat(*m3, *m4);
   ////PrintMat(*m3);
   //DestroyMat(m1);
   //DestroyMat(m2);
   //DestroyMat(m3);
   //DestroyMat(m4);
  /* hipDeviceProp_t prop;
   hipGetDeviceProperties(&prop, 0);
   printf("%d %d %d %d %d\n", prop.sharedMemPerMultiprocessor, prop.sharedMemPerBlock, prop.regsPerMultiprocessor, prop.multiProcessorCount, prop.maxBlocksPerMultiProcessor);*/
   int times = 12;
   float* record = (float*)malloc(sizeof(float) * times * 2);
   CompareSGEMM(record, times);
   for (int i = 0; i < times; i++)
       printf("GFLOPS: SELF: %f, CUBLASS: %f\n", *(record + i), *(record + i + times));

   CHECK_CUDA(hipDeviceReset());
    return 0;
}

__host__ float MulMat(const MATRIX m1, const MATRIX m2, MATRIX* m3, int index) {
    hipEvent_t start, stop;
    float alpha = 1, beta = 0;
    float elapseTime;
    MATRIX device_m1 = MATRIX(), device_m2 = MATRIX(), device_m3 = MATRIX();

    HostToDeviceMat(m1, &device_m1);
    HostToDeviceMat(m2, &device_m2);
    HostToDeviceMat(*m3, &device_m3);
    
    if (index == 9) {
        dim3 gridDim(device_m1.height >> 7, device_m2.width >> 7);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelXI << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else if (index == 8) {
        dim3 gridDim(device_m1.height >> 6, device_m2.width >> 6);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        
        //texture<float4, hipTextureType1D, hipReadModeElementType> texRefA;
        //texture<float4, hipTextureType1D, hipReadModeElementType> texRefB;
        //hipChannelFormatDesc channelDescA = hipCreateChannelDesc<float4>();
        //hipChannelFormatDesc channelDescB = hipCreateChannelDesc<float4>();
        //size_t offsetB;
        //size_t offsetA;
        //hipBindTexture(&offsetA, &texRefA, device_m1.devPtr, &channelDescA, sizeof(device_m1.devPtr));
        //hipBindTexture(&offsetB, &texRefB, device_m2.devPtr, &channelDescB, sizeof(device_m2.devPtr));
        struct hipResourceDesc resDescA;
        resDescA.resType = hipResourceTypeLinear;
        resDescA.res.linear.devPtr = device_m1.devPtr;
        resDescA.res.linear.sizeInBytes = sizeof(float) * device_m1.width * device_m1.height;
        resDescA.res.linear.desc = hipCreateChannelDesc<float4>();
        struct hipTextureDesc texDescA = {};
        texDescA.readMode = hipReadModeElementType;
        hipTextureObject_t texA;
        hipCreateTextureObject(&texA, &resDescA, &texDescA, NULL);
        struct hipResourceDesc resDescB;
        resDescB.resType = hipResourceTypeLinear;
        resDescB.res.linear.devPtr = device_m2.devPtr;
        resDescB.res.linear.sizeInBytes = sizeof(float) * device_m2.width * device_m2.height;
        resDescB.res.linear.desc = hipCreateChannelDesc<float4>();
        struct hipTextureDesc texDescB = {};
        texDescB.readMode = hipReadModeElementType;
        hipTextureObject_t texB;
        hipCreateTextureObject(&texB, &resDescB, &texDescB, NULL);
        hipDeviceSynchronize();
        mulMatrixKernelTV << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, texA, texB, beta, device_m3.devPtr);
        //mulMatrixKernelX << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        //mulMatrixKernelSV << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else if (index == 0) {
        dim3 gridDim(device_m1.height >> 6, device_m2.width >> 6);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelV0 << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else if (index == 1) {
        dim3 gridDim(device_m1.height >> 6, device_m2.width >> 6);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelV1 << <gridDim, 128 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else if (index == 2) {
        dim3 gridDim(device_m1.height >> 6, device_m2.width >> 6);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelV2 << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else if (index == 3) {
        dim3 gridDim(device_m1.height >> 6, device_m2.width >> 6);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelV3 << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else if (index == 4) {
        dim3 gridDim(device_m1.height >> 5, device_m2.width >> 5);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelV4 << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else if (index == 5) {
        dim3 gridDim(device_m1.height >> 6, device_m2.width >> 6);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelV5 << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else if (index == 6) {
        dim3 gridDim(device_m1.height >> 7, device_m2.width >> 7);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelV6 << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }
    else {
        dim3 gridDim(device_m1.height >> 7, device_m2.width >> 7);
        CHECK_CUDA(hipEventCreate(&start));
        CHECK_CUDA(hipEventCreate(&stop));
        CHECK_CUDA(hipEventRecord(start, 0));
        hipDeviceSynchronize();
        mulMatrixKernelV7 << <gridDim, 256 >> > (device_m3.height, device_m3.width, device_m1.width, alpha, device_m1.devPtr, device_m2.devPtr, beta, device_m3.devPtr);
        hipDeviceSynchronize();
        CHECK_CUDA(hipEventRecord(stop, 0));
        CHECK_CUDA(hipEventSynchronize(start));
        CHECK_CUDA(hipEventSynchronize(stop));
    }

    CHECK_CUDA(hipEventElapsedTime(&elapseTime, start, stop));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(device_m1.devPtr));
    CHECK_CUDA(hipFree(device_m2.devPtr));
    CHECK_CUDA(hipMemcpy(m3->devPtr, device_m3.devPtr, sizeof(float) * (m3->width * m3->height), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(device_m3.devPtr));
    printf("CUDA KERNEL %d MULTIMATRIX ELAPSETIME : %f\n", index, elapseTime);

    return elapseTime;
}

__host__ float CublasMM(const MATRIX m1, const MATRIX m2, MATRIX* m3) {
    hipEvent_t start, stop;
    float elapseTime;
    hipblasHandle_t err; hipblasCreate(&err);
    float alpha = 1, beta = 0;
    MATRIX device_m1 = MATRIX(), device_m2 = MATRIX(), device_m3 = MATRIX();
    HostToDeviceMat(m1, &device_m1);
    HostToDeviceMat(m2, &device_m2);
    HostToDeviceMat(*m3, &device_m3);

    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    hipDeviceSynchronize();
    hipblasSgemm(err, HIPBLAS_OP_N, HIPBLAS_OP_N, device_m3.height, device_m3.width, device_m1.width, &alpha, device_m1.devPtr, device_m1.height, device_m2.devPtr, device_m2.height, &beta, device_m3.devPtr, device_m3.height);
    hipDeviceSynchronize();

    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(start));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&elapseTime, start, stop));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(device_m1.devPtr));
    CHECK_CUDA(hipFree(device_m2.devPtr));
    CHECK_CUDA(hipMemcpy(m3->devPtr, device_m3.devPtr, sizeof(float) * (m3->width * m3->height), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(device_m3.devPtr));
    printf("CUDA CUBLAS MULTIMATRIX ELAPSETIME : %f\n", elapseTime);
    return elapseTime;
}

__host__ void HostToDeviceMat(const MATRIX src, MATRIX* dst) {
    dst->width = src.width;
    dst->height = src.height;
    int size = dst->width * dst->height;
    CHECK_CUDA(hipMalloc(&dst->devPtr, sizeof(float) * size));
    CHECK_CUDA(hipMemcpy(dst->devPtr, src.devPtr, sizeof(float) * size, hipMemcpyHostToDevice));
}

__host__ void CompareSGEMM(float* record, int times) {
    int width = 512, height = 512;
    float elapsetime1, elapsetime2, elapsetime3;
    MATRIX* m1 = (MATRIX*)malloc(sizeof(MATRIX)),
        * m2 = (MATRIX*)malloc(sizeof(MATRIX)),
        * m3 = (MATRIX*)malloc(sizeof(MATRIX)),
        * m4 = (MATRIX*)malloc(sizeof(MATRIX));

    for (int i = 0; i < times; i++) {
        width += 512;
        height += 512;
        printf("Test Matrix : WIDTH: %d HEIGHT: %d\n", width, height);
        InitMat(m1, width, height);
        InitMat(m2, height, width);
        InitMat(m3, height, height);
        InitMat(m4, height, height);

        SetMat(m3, 0);
        SetMat(m4, 0);
        HostRANDSetMat(m1, 1);
        HostRANDSetMat(m2, 2);

        warmup << <1, 1 >> > ();

        hipDeviceSynchronize();
        elapsetime1 = MulMat(*m1, *m2, m3, 9);
        hipDeviceSynchronize();
        elapsetime3 = CublasMM(*m1, *m2, m4);
        elapsetime2 = MulMat(*m1, *m2, m3, 7);
        VerifyMat(*m3, *m4);

        DestroyMat(m1);
        DestroyMat(m2);
        DestroyMat(m3);
        DestroyMat(m4);

        *(record + i) = (double)2 * (1e-9) * width * width * height / (elapsetime1 / 1000);
        *(record + i + times) = (double)2 * (1e-9) * width * width * height / (elapsetime3 / 1000);
    }

}
